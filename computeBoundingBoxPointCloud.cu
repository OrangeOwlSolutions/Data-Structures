#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/random.h>
#include <thrust/extrema.h>

/***********************/
/* BOUNDING BOX STRUCT */
/***********************/
struct bbox
{
	float3 lower_left, upper_right;

	// --- Empty box constructor
	__host__ __device__ bbox() {}

	// --- Construct a box from a single point
	__host__ __device__ bbox(const float3 &point) : lower_left(point), upper_right(point) {}

	// --- Construct a box from a pair of points
	__host__ __device__	bbox(const float3 &ll, const float3 &ur) : lower_left(ll), upper_right(ur) {}

};

/*********************************/
/* BOUNDING BOX REDUCTION STRUCT */
/*********************************/
// --- Reduce a pair of bounding boxes (a, b) to a bounding box containing a and b
struct bbox_reduction : public thrust::binary_function<bbox, bbox, bbox>
{
	__host__ __device__ bbox operator()(bbox a, bbox b)
	{
		// --- Lower left corner
		float3 ll = make_float3(thrust::min(a.lower_left.x, b.lower_left.x), thrust::min(a.lower_left.y, b.lower_left.y), thrust::min(a.lower_left.z, b.lower_left.z));

		// --- Upper right corner
		float3 ur = make_float3(thrust::max(a.upper_right.x, b.upper_right.x), thrust::max(a.upper_right.y, b.upper_right.y), thrust::max(a.upper_right.z, b.upper_right.z));

		return bbox(ll, ur);
	}
};

/********/
/* MAIN */
/********/
int main(void)
{
	const size_t N = 40;
	thrust::default_random_engine rng;
	thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

	// --- Allocate space for 3D points
	thrust::device_vector<float3> d_points(N);

	// --- Generate random 3D points in the unit cube
	for (size_t i = 0; i < N; i++)
	{
		float x = u01(rng);
		float y = u01(rng);
		float z = u01(rng);
		d_points[i] = make_float3(x, y, z);
	}

	// --- The initial bounding box contains the first point of the point cloud
	bbox init = bbox(d_points[0], d_points[0]);

	// --- Binary reduction operation
	bbox_reduction binary_op;

	// --- Compute the bounding box for the point set
	bbox result = thrust::reduce(d_points.begin(), d_points.end(), init, binary_op);

	for (int k = 0; k < N; k++) {
		float3 temp = d_points[k];
		printf("%d %f %f %f\n", k, temp.x, temp.y, temp.z);
	}
	
	// --- Print output
	std::cout << "bounding box " << std::fixed;
	std::cout << "(" << result.lower_left.x << "," << result.lower_left.y << "," << result.lower_left.z << ") ";
	std::cout << "(" << result.upper_right.x << "," << result.upper_right.y << "," << result.upper_right.z << ")" << std::endl;

	return 0;
}
